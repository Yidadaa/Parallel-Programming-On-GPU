#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include "check.h"

#define SOFTENING 1e-9f

/*
 * Each body contains x, y, and z coordinate positions,
 * as well as velocities in the x, y, and z directions.
 */

typedef struct { float x, y, z, vx, vy, vz; } Body;

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

void randomizeBodies(float *data, int n) {
  for (int i = 0; i < n; i++) {
    data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
  }
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

__global__
void bodyForce(Body *p, Body *tmp, float dt, int n) {
  int xindex = threadIdx.x + blockIdx.x * blockDim.x;
  int xstride = blockDim.x * gridDim.x;

  for(int i = xindex; i < n; i += xstride)
  {
    float Fx = 0.0f; float Fy = 0.0f; float Fz = 0.0f;

    for (int j = 0; j < n; j++) {
      float dx = p[j].x - p[i].x;
      float dy = p[j].y - p[i].y;
      float dz = p[j].z - p[i].z;

      float distSqr = dx*dx + dy*dy + dz*dz + SOFTENING;
      float invDist = rsqrtf(distSqr);
      float invDist3 = invDist * invDist * invDist;

      Fx += dx * invDist3; Fy += dy * invDist3; Fz += dz * invDist3;
    }
    p[i].vx += dt*Fx; p[i].vy += dt*Fy; p[i].vz += dt*Fz;
    tmp[i].vx = p[i].vx;
    tmp[i].vy = p[i].vy;
    tmp[i].vz = p[i].vz;
    tmp[i].x = p[i].x + p[i].vx*dt;
    tmp[i].y = p[i].y + p[i].vy*dt;
    tmp[i].z = p[i].z + p[i].vz*dt;
  }
}

int main(const int argc, const char** argv) {

  int deviceId;
  int numberOfSMs;
  hipGetDevice(&deviceId);
  hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
  
  size_t nthreads = 512;
  size_t nblocks = 32 * numberOfSMs;
  
  hipError_t addVectorsErr;
  hipError_t asyncErr;

  /*
   * Do not change the value for `nBodies` here. If you would like to modify it,
   * pass values into the command line.
   */

  int nBodies = 2<<11;
  int salt = 0;
  if (argc > 1) nBodies = 2<<atoi(argv[1]);

  /*
   * This salt is for assessment reasons. Tampering with it will result in automatic failure.
   */

  if (argc > 2) salt = atoi(argv[2]);

  const float dt = 0.01f; // time step
  const int nIters = 10;  // simulation iterations

  int bytes = nBodies * sizeof(Body);
  
  float *host_buf;
  float *device_buf_1;
  float *device_buf_2;
  
  hipMalloc(&device_buf_1, bytes);
  hipMalloc(&device_buf_2, bytes);
  hipHostMalloc(&host_buf, bytes);

  /*
   * As a constraint of this exercise, `randomizeBodies` must remain a host function.
   */

  randomizeBodies(host_buf, 6 * nBodies); // Init pos / vel data
  
  hipMemcpy(device_buf_1, host_buf, bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_buf_2, host_buf, bytes, hipMemcpyHostToDevice);

  double totalTime = 0.0;
  
  int flag = 0;

  /*
   * This simulation will run for 10 cycles of time, calculating gravitational
   * interaction amongst bodies, and adjusting their positions to reflect.
   */

  /*******************************************************************/
  // Do not modify these 2 lines of code.
  for (int iter = 0; iter < nIters; iter++) {
      StartTimer();
      /*******************************************************************/

      /*
       * You will likely wish to refactor the work being done in `bodyForce`,
       * as well as the work to integrate the positions.
       */
      flag = 1 - flag;
      if (flag) {
          bodyForce<<<nblocks, nthreads>>>((Body*)device_buf_1, (Body*)device_buf_2, dt, nBodies);
      } else {
          bodyForce<<<nblocks, nthreads>>>((Body*)device_buf_2, (Body*)device_buf_1, dt, nBodies);
      }

      /*******************************************************************/
      // Do not modify the code in this section.
      const double tElapsed = GetTimer() / 1000.0;
      totalTime += tElapsed;
  }
  if (flag) {
      hipMemcpy(host_buf, device_buf_2, bytes, hipMemcpyHostToDevice);
  } else {
      hipMemcpy(host_buf, device_buf_1, bytes, hipMemcpyHostToDevice);
  }
  
  addVectorsErr = hipGetLastError();
  if(addVectorsErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(addVectorsErr));

  asyncErr = hipDeviceSynchronize();
  if(asyncErr != hipSuccess) printf("Error: %s\n", hipGetErrorString(asyncErr));

  double avgTime = totalTime / (double)(nIters);
  float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
  checkPerformance(host_buf, billionsOfOpsPerSecond, salt);
#else
  checkAccuracy(host_buf, nBodies);
  printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
  salt += 1;
#endif
  /*******************************************************************/

  /*
   * Feel free to modify code below.
   */

  hipFree(device_buf_1);
  hipFree(device_buf_2);
  hipHostFree(host_buf);
}
